#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define Tile_size 16
int form_matrix(char input[],double mat[][1000],int *i,int *j,bool test)
{
	int l=0,k=0;
	char temp[100000];
	int number_of_cols=0;
	while(input[k]!='\n')
    {  		
    	if(input[k]!=' ')
	    {
	   		temp[l]=input[k];
	   		l++;
	   		k++;
	   	}
    	else if (input[k]==' ') 
    	{
    		for (int z=l;z<100000;z++)
    			temp[z]=0;
        //printf("%s\n",temp);
    		mat[*i][*j]=atof(temp);
    		strcpy(temp," ");
    		l=0;
    		k++;
    		(*j)++;
    		number_of_cols++;   		
    	}
    }
   	if (input[k]=='\n' && test!=true)
   	{
   		for (int z=l;z<100000;z++)
    			temp[z]=0;
      //printf("%s\n",temp);
   		mat[*i][*j]=atof(temp);
    	strcpy(temp," ");
    	l=0;
    	k=0;
    	(*i)++;
    	*j=0;
    	number_of_cols++;
   	}
   	input[0]='\0';
   	return number_of_cols;
}
__global__ void
convolution(const double *a, const double *h, double *c,int c_rows, int c_cols,int z,int i,int j,int k)
{
  int n = blockIdx.x * Tile_size + threadIdx.x; //idx
  int m = blockIdx.y * Tile_size + threadIdx.y; //idy
  //for(int x=0,y=0;x<c_cols;x+=Tile_size,y+=Tile_size)
  {
    __shared__ double H[Tile_size][Tile_size];
    __shared__ double A[Tile_size][Tile_size];
    if (m>(Tile_size-1)||n>(Tile_size-1))
    {
     
      if((m%Tile_size)<j && (n%Tile_size)<k)
      H[m%Tile_size][n%Tile_size]=h[(k*(m%Tile_size)+(n%Tile_size))];
      __syncthreads();
      if((m%Tile_size)<i && (n%Tile_size)<z)
      {
        A[m%Tile_size][n%Tile_size]=a[(i*(m%Tile_size)+(n%Tile_size))];
      __syncthreads();
      }

    }else
    {
      if(m<j && n<k)
        H[m][n]=h[(k*m)+n];
  	   __syncthreads();
      if(m<i && n<z)
        A[m][n]=a[(i*m)+n];
      __syncthreads();
    }
    if (m<c_rows && n<c_cols)
    {
      for(int p=0;p<=(Tile_size-1);p++)
      {
        for(int q=0;q<=(Tile_size-1);q++)
        {
          if(!((m-p)<0 || (n-q)<0 || (m-p)>=z || (n-q)>=i))
          {
            c[(m*c_cols)+n]+=H[p][q]*A[(m-p)][(n-q)];     
            __syncthreads();
          }
        }
      }
    }
  }
}
int main(int argc, char **argv)
{
	FILE *read_file;
	char input[100000];
	int e=0,d=0,m=0,k=0,j=0,n=0,select=1,u=0,v=0;
	double a[1000][1000],h[20][1000];
  hipError_t err = hipSuccess;
	int flag1=0,flag2=0;
  char *input_file;
  input_file=argv[1];
	read_file=fopen(input_file,"r");
	if (read_file==NULL)
	{
		printf("Error opening file\n");
		exit(1);
	}
	while(fgets(input,100000,read_file)) 
    {
     	bool test=false;
    	if (strcmp(input,"\n")==0)
    	{
    		select=2;
     		test=true;
    	}
    	if (select==1)
    	{
    		(m)++;
        if (test!=true && flag1==0)
        {
    		  n=form_matrix(input,a,&e,&d,test);
          flag1=1;
        }
        else
          form_matrix(input,a,&e,&d,test);
    	}
    	else if (select==2)
    	{
    		(j)++;
    		if (test!=true && flag2==0)
    		{
    			k=form_matrix(input,h,&u,&v,test);
    			flag2=1;
    		}
    		else 
    			form_matrix(input,h,&u,&v,test);
    	}
    	input[0]='\0';
    }
    --j;
    /*printf("Size of matrix 1:%d * %d\n",m,n);
    printf("Size of matrix 2:%d * %d\n",j,k);
   	for (int i=0;i<5;i++)
   	{
   		for(int j=0;j<5;j++)
   		{
   			printf("%f ",a[i][j]);
   		}
   		printf("\n");
   	}
   	for (int i=0;i<j;i++)
   	{
   		for(int z=0;z<k;z++)
   		{
   			printf("%f ",h[i][z]);
   		}
   		printf("\n");
   	}*/
    size_t size_a=(m*n)*sizeof(double);
    double *h_a=(double*)malloc(size_a);
    for (int i=0;i<m;i++)
    {
      for(int j=0;j<n;j++)
      {
        h_a[(i*n)+j]=a[i][j];
      }
    }
   	double *d_a=NULL;
   	err = hipMalloc((void **)&d_a, size_a);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    size_t size_h=(j*k)*sizeof(double);
    double *h_h=(double*)malloc(size_h);
    for (int p=0;p<j;p++)
    {
      for(int q=0;q<k;q++)
      {
        h_h[(p*k)+q]=h[p][q];
      }
    }
    double *d_h=NULL;
   	err = hipMalloc((void **)&d_h, size_h);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix h (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int c_rows=(m+j-1);
    int c_cols=(n+k-1);
	  double *d_c=NULL;
   	size_t size_c=(c_rows*c_cols)*sizeof(double);
    double *c=(double*)malloc(size_c);
   	err = hipMalloc((void **)&d_c, size_c);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix c (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix a from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_h, h_h, size_h, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix h from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //dim3 threadsPerBlock(32,32);
    dim3 threadsPerBlock(Tile_size,Tile_size);
    dim3 numBlocks(c_rows/threadsPerBlock.x+1,c_cols/threadsPerBlock.y+1);
    struct timeval begin, end;
    gettimeofday(&begin, NULL);
    convolution<<<numBlocks, threadsPerBlock>>>(d_a, d_h, d_c,c_rows, c_cols, m, n, j, k);
    gettimeofday(&end, NULL);
  
    int time_in_us = 1e6*(end.tv_sec-begin.tv_sec) + (end.tv_usec-begin.tv_usec);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   err = hipDeviceSynchronize();
  
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize the device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix c from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   	
   	for (int i=0;i<(m+j-1);i++)
   	{
   		for(int z=0;z<(n+k-1);z++)
   		{
   			printf("%0.3lf ",c[(i*c_cols)+z]);
   		}
   		printf("\n");
   	}
    printf("Time for V1 Kernel = %d us\n", time_in_us);
   	err = hipFree(d_a);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_h);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix h (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_c);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix c (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    free(h_a);
    free(h_h);
    free(c);
    fclose(read_file);
	return 0;
}